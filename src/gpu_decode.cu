#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include "binary_tree.h"

using namespace std;

extern __constant__ char d_tree_arr_const[];

__device__ void append(char *s, char c, int position)
{
	s[position] = c;
	s[position +1] = '\0';
}

__device__ int string_count = 0;

__global__
void decode_kernel(int k, int count)
{
  /*if(threadIdx.x == 0) {
    hipStream_t s1, s2;
    //unsigned int flag = hipStreamDefault;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
    int l = 2*k+1;
    int r = 2*k+2;
    if (d_tree_arr[l] == '$' && d_tree_arr[r] == '$')
    {
        //printf("\nDecoded CHAR:%c",d_tree_arr[k]);
        string_count = count;
    }
    if (encoded_str[count] == '0' && d_tree_arr[l] != '$' ){
      decode_kernel<<<1,1,0, s1>>>(l, count + 1);
      hipDeviceSynchronize();
    }

    if (encoded_str[count] == '1' && d_tree_arr[r] != '$') {
      //printf("got right 1 count:%d\n", count);
      //count++;

      decode_kernel<<<1,1,0, s1>>>( r, count + 1);
      hipDeviceSynchronize();
      //hipStreamDestroy(s2);
    }
  }
  __syncthreads();*/
}

__global__
void decode_parent_kernel(int string_size) {

  if(threadIdx.x == 0){
    while (string_count < string_size) {
      decode_kernel<<<1,1,0>>>(0, string_count);
      hipDeviceSynchronize();
    }
  }
  __syncthreads();
}

void gpu_decode(int input_str_length) {
	hipError_t err = hipSuccess;
    char h_final_str[input_str_length];
    char *d_final_str = NULL;
    err = hipMalloc((void**)&d_final_str, input_str_length* sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device input string (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

  	hipEventRecord(start);
    int blocksPerGrid = 8; //(input_str_length / 1024) + 1;
    int threadsPerBlock = 32; //1024;// FILL HERE
    decode_parent_kernel<<<blocksPerGrid, threadsPerBlock>>>(input_str_length);
  	hipEventRecord(stop);

  	hipEventSynchronize(stop);

    float elapsed = 0;
  	hipEventElapsedTime(&elapsed, start, stop);
  	printf("The elapsed time for Decode kernal excution is %.2f ms\n", elapsed);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipDeviceSynchronize();

    err = hipMemcpy(h_final_str, d_final_str, input_str_length, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy final string from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
    //hipFree(d_str);
    //hipFree(d_encode_map);
    //hipFree(d_input_string);
    hipFree(d_final_str);
}

void cpu_decode(Node* root, int &index, string str)
{
    if (root == nullptr) {
        return;
    }

    // found a leaf node
    if (!root->left && !root->right)
    {
        cout << root->ch;
        return;
    }

    index++;

    if (str[index] =='0')
        cpu_decode(root->left, index, str);
    else
        cpu_decode(root->right, index, str);
}
