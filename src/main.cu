#include "huffman.h"
#include "histogram.h"

// Huffman coding algorithm
int main()
{
	string text = "Huffman coding is a data compression algorithm.";
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	buildHuffmanTree(text);

	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	cout << "The elapsed time for Algorithm excution is : " << elapsed;

	hipEventDestroy(start);
	hipEventDestroy(stop);
    return 0;
}
