#include <fstream>
#include <sstream>
#include "huffman.h"
#include "histogram.h"

// Huffman coding algorithm
int main()
{
	//string text = "Huffman coding is a data compression algorithm.";
	std::string filename = "SampleTextFile_10kb.txt";
 	std::ifstream file(filename.c_str());
	std::stringstream buffer;
  buffer << file.rdbuf();
  std::string text = buffer.str();
	//string text = "Huffman";
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	buildHuffmanTree(text);

	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The elapsed time for Algorithm excution is %.2f ms\n", elapsed);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	//hipDeviceSynchronize();
    return 0;
}
