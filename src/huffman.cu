#include "hip/hip_runtime.h"
#include "huffman.h"
#include "histogram.h"
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define TOTAL_CHARS 256
#define MEM_WIDTH 256*8
#define MAX_CODE_WIDTH 8 //pow(2, height(root)); //Worst case height of tree when all chars have same frequency
#define MAX_NODES 511//(pow(2, 9) - 1)


// Comparison object to be used to order the heap
struct comp
{
    bool operator()(Node* l, Node* r)
    {
        // highest priority item has lowest frequency
        return l->freq > r->freq;
    }
};

// traverse the Huffman Tree and store Huffman Codes
// in a map.
void encode(Node* root, string str, unordered_map<char, string> &huffmanCode)
{
    if (root == nullptr)
        return;

    // found a leaf node
    if (!root->left && !root->right) {
        huffmanCode[root->ch] = str;
    }

    encode(root->left, str + "0", huffmanCode);
    encode(root->right, str + "1", huffmanCode);
}

// traverse the Huffman Tree and decode the encoded string
void decode(Node* root, int &index, string str)
{
    if (root == nullptr) {
        return;
    }

    // found a leaf node
    if (!root->left && !root->right)
    {
        cout << root->ch;
        return;
    }

    index++;

    if (str[index] =='0')
        decode(root->left, index, str);
    else
        decode(root->right, index, str);
}

//__device__ char* device_encode_map[TOTAL_CHARS * MAX_CODE_WIDTH];
__constant__ char d_tree_arr [MAX_NODES];
__global__ void encode_kernel(int k, int count,char *str,  char *encode_map)
{
  if(threadIdx.x == 0){
    hipStream_t s1, s2;
    //unsigned int flag = hipStreamDefault;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
    //  printf("\nInside Kernel : %d and count : %d and string : %s",k, count, str);
    int l = 2*k+1;
    int r = 2*k+2;
    //__shared__ char s_str[MAX_CODE_WIDTH];
    if (d_tree_arr[l] == '$' && d_tree_arr[r] == '$')
    {
      //s_str = str[];
      int i = 0;
      #pragma unroll
      for ( ;i < MAX_CODE_WIDTH && str[i] != '\0'; i++) {
          encode_map[d_tree_arr[k]*MAX_CODE_WIDTH + i] = str[i];

      }
      encode_map[d_tree_arr[k]*MAX_CODE_WIDTH + i+1] = '\0';

		  // printf("\nCHAR:%c ENCODEDING:%s",d_tree_arr[k], str);
      //fill_Map <<<1,1,0, s1>>>
	  }
    //printf("\nCHAR:%c", d_tree_arr[k]);
    if (d_tree_arr[l] != '$'){
      //if(count != -1)
      str[count] = '0';
      str[count+1] = '\0';
      //append(str,'0', count);
      //printf("Appending: 0");
      //  printf("\nCHAR:%c LEFT ENCODEDING:%s",d_tree_arr[k], str);
      encode_kernel<<<1,1,0, s1>>>(l,count + 1, str, encode_map);
      //append(str,'\0',count);
      hipDeviceSynchronize();
    }

    if (d_tree_arr[r] != '$') {
      //  printf("Appending: 1");
      //  if(count != -1)
      str[count] = '1';
      str[count+1] = '\0';
      //append(str,'1',count);
      //printf("\nCHAR:%c RIGHT ENCODEDING:%s",d_tree_arr[k], str);

      encode_kernel<<<1,1,0, s2>>>(r, count + 1, str, encode_map);
      hipDeviceSynchronize();
    }
  }
  __syncthreads();
}
__device__ int string_count = 0;



__global__
void decode_kernel(char *encoded_str,int k, int count) {
  if(threadIdx.x == 0){
    hipStream_t s1, s2;
    //unsigned int flag = hipStreamDefault;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
    int l = 2*k+1;
    int r = 2*k+2;
    if (d_tree_arr[l] == '$' && d_tree_arr[r] == '$')
    {
        printf("\nDecoded CHAR:%c",d_tree_arr[k]);
        string_count = count;
    }
    if (encoded_str[count] == '0' && d_tree_arr[l] != '$' ){
      decode_kernel<<<1,1,0, s1>>>(encoded_str, l, count + 1);
      hipDeviceSynchronize();
    }

    if (encoded_str[count] == '1' && d_tree_arr[r] != '$') {
      //printf("got right 1 count:%d\n", count);
      //count++;

      decode_kernel<<<1,1,0, s2>>>(encoded_str, r, count + 1);
      hipDeviceSynchronize();
      //hipStreamDestroy(s2);
    }
  }
  __syncthreads();
}

/*__device__ void test(int cont){
  printf("Test%d\n", count);
  count++;
}*/
__global__
void decode_parent_kernel(char *encoded_str, int string_size) {

  if(threadIdx.x == 0){
    while (string_count < string_size) {
      decode_kernel<<<1,1,0>>>(encoded_str,0, string_count);
      hipDeviceSynchronize();
    }
  }
  __syncthreads();
}
__global__ void my_concat_kernel(char *encode_table, char *input_string, char *encoded_string, size_t input_size)
{
    //printf("In encode_kernel input string:%c\n", input_string[threadIdx.x]);
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    //printf("tid:%d\n", tid);
    //int row = tid / MAX_CODE_WIDTH;
    //int col = tid % MAX_CODE_WIDTH;
    if (tid < input_size) {
        for (int i = 0; i < MAX_CODE_WIDTH; i++) {
            encoded_string[tid * MAX_CODE_WIDTH + i] = encode_table[input_string[tid] * MAX_CODE_WIDTH + i];
        }
    }
    __syncthreads();
}

void generateEncodedString(unordered_map<char, string> huffmanCode, string &text, string &final_encoded_string)
{
    hipError_t err = hipSuccess;
    int table_size = TOTAL_CHARS * MAX_CODE_WIDTH * sizeof(char);
    //cout << "table size: " << table_size << endl;
    char *h_encode_table = (char*) malloc(table_size);
    int encode_string_size = text.length() * MAX_CODE_WIDTH * sizeof(char);
    char *h_encoded_string = (char*) malloc(encode_string_size);
    if (h_encode_table == NULL || h_encoded_string == NULL) {
        fprintf(stderr, "Failed to allocate host table or string (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    memset(h_encode_table, 0, table_size);
    memset(h_encoded_string, 0, encode_string_size);

    //cout << "Huffman Codes are :\n" << '\n';
    for (auto pair: huffmanCode) {
        //cout << pair.first << " " << pair.second << '\n';
        for (int j = 0; j < MAX_CODE_WIDTH; j++) {
            if (j < pair.second.length())
                h_encode_table[pair.first * MAX_CODE_WIDTH + j] = pair.second[j];
        }
        //cout << endl;
    }

    // Allocate encode kernel variables
    char h_input_string[text.length()];
    strcpy(h_input_string, text.c_str());
    //cout << "input text:" << text << endl;
    //cout << "input string in host array:" << h_input_string << endl;
    char *d_input_string = NULL;
    err = hipMalloc((void**)&d_input_string, text.length()* sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device input string (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_input_string, h_input_string, text.length() * sizeof(char),hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input string from host to device (error code %s)!\n", 	  hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    char *d_encode_table = NULL;
    err = hipMalloc((void**)&d_encode_table, table_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device encode table (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_encode_table, h_encode_table, table_size,hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy encode table from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    char *d_encoded_string = NULL;
    err = hipMalloc((void**)&d_encoded_string, encode_string_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device encode string (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int blocksPerGrid = (text.length() / 1024) + 1;
    int threadsPerBlock = 1024;// FILL HERE
	printf("CUDA encode kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

	hipEventRecord(start);
    //printf("Before my_concat_kernel");
    my_concat_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_encode_table, d_input_string, d_encoded_string, text.length());
    //printf("After my_concat_kernel");
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The elapsed time for encode kernal exexution is %.2f ms\n", elapsed);

    hipEventDestroy (start);
    hipEventDestroy (stop);
    hipDeviceSynchronize();

    err = hipMemcpy(h_encoded_string, d_encoded_string, encode_string_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy encoded string from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    printf("After kernel execution result:\n");
    for(int i = 0; i < text.length()*8; i++) {
        printf("%c", h_encoded_string[i]);
        if (h_encoded_string[i] != 0)
            final_encoded_string += h_encoded_string[i];
    }
    // Free Device variables
    hipFree(d_input_string);
    hipFree(d_encode_table);
    hipFree(d_encoded_string);

    // Free host memory
    free(h_encode_table);
    free(h_encoded_string);

}

// Builds Huffman Tree and decode given input text
void buildHuffmanTree(string text)
{
    hipError_t err = hipSuccess;
    int nodes = 0;
    // count frequency of appearance of each character and store it in a map
    unordered_map<char, int> freq;
    calculateFrequencies(text, freq);
    // Create a priority queue to store live nodes of
    // Huffman tree;
    priority_queue<Node*, vector<Node*>, comp> pq;

    // Create a leaf node for each character and add it
    // to the priority queue.
    for (auto pair: freq) {
        pq.push(getNode(pair.first, pair.second, nullptr, nullptr));
        nodes++;
    }

    // do till there is more than one node in the queue
    while (pq.size() != 1)
    {
        // Remove the two nodes of highest priority
        // (lowest frequency) from the queue
        Node *left = pq.top(); pq.pop();
        Node *right = pq.top();    pq.pop();

        // Create a new internal node with these two nodes
        // as children and with frequency equal to the sum
        // of the two nodes' frequencies. Add the new node
        // to the priority queue.
        int sum = left->freq + right->freq;
        pq.push(getNode('\0', sum, left, right));
        nodes++;
    }

    //int inx = 0;
    //preorder(pq, a, &inx);
    printf("Total nodes:%d\n", nodes);
    // root stores pointer to root of Huffman Tree
    Node* root = pq.top();
    int treeHeight = height(root);
    char *h_arr = (char *)malloc(sizeof(char)*MAX_NODES);
    memset(h_arr, '\0', MAX_NODES);
    printLevelOrder(root, h_arr, treeHeight);
    printf("Tree converted to array :\n");
    for (int i = 0; i < 15; i++){
      //  h_arr[i] = 65 + i;
        printf("%c->", h_arr[i]);
    }

    char *d_str = NULL;
    err = hipMalloc((void**)&d_str, MAX_CODE_WIDTH * sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device string(str) (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMemset(d_str, '\0', MAX_CODE_WIDTH);

    err = hipMemcpyToSymbol(HIP_SYMBOL(d_tree_arr), h_arr, MAX_NODES*sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy tree array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int table_size = TOTAL_CHARS * MAX_CODE_WIDTH * sizeof(char);
    char *h_encode_map = (char*) malloc(table_size);
    if (h_encode_map == NULL)
    {
        fprintf(stderr, "Failed to allocate host encode map (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    memset(h_encode_map, 0, table_size);

    char *d_encode_map = NULL;
    err = hipMalloc((void**)&d_encode_map, table_size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device encode map (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    unordered_map<char, string> huffmanCode;
    //encode(root, "", huffmanCode);
    int blocksPerGrid = 1;//(text.length() / 1024) + 1;
    int threadsPerBlock = 1;//1024;// FILL HERE
    printf("CUDA encode kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 9);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

  	hipEventRecord(start);
    encode_kernel<<<blocksPerGrid, threadsPerBlock>>>(0, 0, d_str, d_encode_map);
  	hipEventRecord(stop);

  	hipEventSynchronize(stop);

  	float elapsed = 0;
  	hipEventElapsedTime(&elapsed, start, stop);
  	printf("The elapsed time for Encode kernal excution is %.2f ms\n", elapsed);

    hipEventDestroy (start);
    hipEventDestroy (stop);
    hipDeviceSynchronize();

    err = hipMemcpy(h_encode_map, d_encode_map, table_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy encode map from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#if 0
    printf("Final Encoded Table\n");
    //printf("\nCHAR:%c ENCODEDING:%s",h_arr[4], h_encode_map[97*MAX_CODE_WIDTH]);
    for (int i = 0; i < MAX_NODES ; i++) {
      if(h_arr[i] != '$' && h_arr[i] != '*' && h_arr[i] != '\0') {
        printf("\nchar:%c:", h_arr[i]);
        for (int j = 0; j < 8; j++) {
          char tmp = h_encode_map[h_arr[i]*MAX_CODE_WIDTH + j];
          if (tmp != '\0')
            printf("%c", tmp);
        }
      }
    }
    printf("\n\n");
#endif

    hipDeviceSynchronize();
    //generateEncodedString(huffmanCode, text, str);
    cout << "\nOriginal string was :\n" << text << '\n';
    // print encoded string
    string str = "";
    for (char ch: text) {
        for (int i = 0; h_encode_map[(ch * MAX_CODE_WIDTH) + i] != '\0'; i++)
            str += h_encode_map[ch * MAX_CODE_WIDTH + i];
    }
    cout << "\nEncoded string is :\n" << str << '\n';


    char h_input_string[str.length()];
    strcpy(h_input_string, str.c_str());
    char *d_input_string = NULL;
    err = hipMalloc((void**)&d_input_string, str.length()* sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device input string (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMemset(d_input_string, '\0', str.length());
    err = hipMemcpy(d_input_string, h_input_string, str.length() * sizeof(char),hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input string from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    char h_final_str[str.length()];
    char *d_final_str = NULL;
    err = hipMalloc((void**)&d_final_str, text.length()* sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device input string (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);

  	hipEventRecord(start);
    decode_parent_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input_string, str.length());
  	hipEventRecord(stop);

  	hipEventSynchronize(stop);

    elapsed = 0;
  	hipEventElapsedTime(&elapsed, start, stop);
  	printf("The elapsed time for Encode kernal excution is %.2f ms\n", elapsed);

    hipEventDestroy (start);
    hipEventDestroy (stop);
    hipDeviceSynchronize();

    err = hipMemcpy(h_final_str, d_final_str, text.length(), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy final string from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
    hipFree(d_str);
    hipFree(d_encode_map);
    hipFree(d_input_string);
    hipFree(d_final_str);
  //  hipFree(d_encode_table);
    //hipFree(d_encoded_string);


    //free(h_encode_table);
    //free(h_encoded_string);
    free(h_arr);
    free(h_encode_map);

}
