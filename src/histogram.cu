#include "hip/hip_runtime.h"
#include "histogram.h"

#define TOTAL_CHARS 256

__global__ void histo_kernel(char *buffer, long size, unsigned int *histo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(i < size) {
		atomicAdd( &(histo[(buffer[i])]), 1);
		i += stride;
	}
	__syncthreads();
}

void calculateFrequencies(char *char_array, int input_str_length, unordered_map<char, int> &freq) {
    //cout << "input:" << text << endl;
    hipError_t err = hipSuccess;

	// Allocate the host input matrix h_A
    int histo_size = TOTAL_CHARS * sizeof(int);
    unsigned int *h_histo = (unsigned int *)malloc(histo_size);

    // Verify that allocations succeeded
    if (h_histo == NULL)
    {
        fprintf(stderr, "Failed to allocate host histograms!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input matrices
    for (int i = 0; i < TOTAL_CHARS; ++i)
    {
      h_histo[i] = 0;
    }
    unsigned int *d_histo =NULL;
    err = hipMalloc((void**)&d_histo, histo_size * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Allocate the device output matrix
    char *d_char_array = NULL;
    err = hipMalloc((void**)&d_char_array, input_str_length* sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_histo, h_histo, histo_size * sizeof(int),hipMemcpyHostToDevice);// FILL HERE
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	err = hipMemcpy(d_char_array, char_array, input_str_length * sizeof(char),hipMemcpyHostToDevice);// FILL HERE
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix B from host to device (error code %s)!\n", 	  hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	int blocksPerGrid = 7;// FILL HERE
    int threadsPerBlock = 128;// FILL HERE
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

	hipEventRecord(start);
	histo_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_char_array, input_str_length, d_histo);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The elapsed time for histogram kernal exexution is %.2f ms\n", elapsed);

    hipEventDestroy (start);
    hipEventDestroy (stop);
    // <--

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matrixMul kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	hipDeviceSynchronize();

    // Copy the device result matrix in device memory to the host result matrix
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_histo, d_histo, histo_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    //Testing
    for(int i = 0; i < TOTAL_CHARS; i++){
		if (h_histo[i] != 0) {
			printf("\nindex: %c   Frequency: %d", i, h_histo[i]);
			freq.insert( std::pair<char,int>(i,h_histo[i]));
		}
    }

    hipFree(d_histo);
    // Free host memory
	free(h_histo);
}
