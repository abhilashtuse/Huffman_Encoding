#include "hip/hip_runtime.h"
#include "histogram.h"

#define TEN_KB 10250
#define TOTAL_CHARS 256

__constant__ char d_input_string_const[TEN_KB];

__global__ void histo_kernel(long size, unsigned int *histo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(i < size) {
		atomicAdd( &(histo[(d_input_string_const[i])]), 1);
		i += stride;
	}
	__syncthreads();
}


void calculateFrequencies(char *char_array, int input_str_length, unordered_map<char, int> &freq) {
    //cout << "input:" << text << endl;
    hipError_t err = hipSuccess;

	// Allocate the host input matrix h_A
    int histo_size = TOTAL_CHARS * sizeof(int);
    unsigned int *h_histo = (unsigned int *)malloc(histo_size);

    // Verify that allocations succeeded
    if (h_histo == NULL)
    {
        fprintf(stderr, "Failed to allocate host histograms!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input matrices
    for (int i = 0; i < TOTAL_CHARS; ++i)
    {
      h_histo[i] = 0;
    }
    unsigned int *d_histo =NULL;
    err = hipMalloc((void**)&d_histo, histo_size * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpyToSymbol(HIP_SYMBOL(d_input_string_const), char_array, input_str_length * sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy input string from host to device (error code %s)!\n", 	  hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	int blocksPerGrid = 7;// FILL HERE
    int threadsPerBlock = 128;// FILL HERE
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

	hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

	hipEventRecord(start);
	histo_kernel<<<blocksPerGrid, threadsPerBlock>>>(input_str_length, d_histo);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	float elapsed = 0;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("The elapsed time for histogram kernal exexution is %.2f ms\n", elapsed);

    hipEventDestroy (start);
    hipEventDestroy (stop);
    // <--

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matrixMul kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	hipDeviceSynchronize();

    // Copy the device result matrix in device memory to the host result matrix
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_histo, d_histo, histo_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    //Testing
    for(int i = 0; i < TOTAL_CHARS; i++){
		if (h_histo[i] != 0) {
			printf("\nindex: %c   Frequency: %d", i, h_histo[i]);
			freq.insert( std::pair<char,int>(i,h_histo[i]));
		}
    }

    hipFree(d_histo);
    // Free host memory
	free(h_histo);
}
